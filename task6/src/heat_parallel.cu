#include "hip/hip_runtime.h"
	#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_cooperative_groups.h>

#include "../inc/argument_utils.h"
namespace cg = cooperative_groups;

// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

typedef int64_t int_t;
typedef double real_t;

int_t
    M,
    N,
    max_iteration,
    snapshot_frequency;

real_t
    dt,
    *h_temp,
    *h_thermal_diffusivity,
    //1: Declare device side pointers to store host-side data.
    *d_temp,
    *d_thermal_diffusivity;

#define T(x,y)                      temp[(y) * (N + 2) + (x)]
#define THERMAL_DIFFUSIVITY(x,y)    thermal_diffusivity[(y) * (N + 2) + (x)]

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void __global__ time_step_kernel ( real_t *temp, real_t *thermal_diffusivity, int_t M, int_t N, real_t dt);
void __device__ boundary_condition (real_t *temp, int_t M, int_t N, int x, int y );
void domain_init ( void );
void domain_save ( int_t iteration );
void domain_finalize ( void );


int
main ( int argc, char **argv )
{
    OPTIONS *options = parse_args( argc, argv );
    if ( !options )
    {
        fprintf( stderr, "Argument parsing failed\n" );
        exit(1);
    }

    M = options->M;
    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;

    domain_init();

    struct timeval t_start, t_end;
    gettimeofday ( &t_start, NULL );

    for ( int_t iteration = 0; iteration <= max_iteration; iteration++ )
    {
        //6: Launch the time-step kernel using cooperative groups syntax

	void *kernelArgs[] = {
		(void*) &d_temp,
		(void*) &d_thermal_diffusivity,
		(void*) &M,
		(void*) &N,
		(void*) &dt,
	};

	dim3 blockSize ( 32, 32, 1 );
        int gridX = (unsigned int) ceil( (double)(N+2)/(double)blockSize.x );
	int gridY = (unsigned int) ceil( (double)(M+2)/(double)blockSize.y );
	dim3 gridSize ( gridX, gridY, 1 );
	hipLaunchCooperativeKernel ( (void*) time_step_kernel, gridSize, blockSize, kernelArgs );
	cudaErrorCheck( hipPeekAtLastError() );

        if ( iteration % snapshot_frequency == 0 )
        {
            printf (
                "Iteration %ld of %ld (%.2lf%% complete)\n",
                iteration,
                max_iteration,
                100.0 * (real_t) iteration / (real_t) max_iteration
            );

            //7: Copy data from device to host.
	    hipMemcpy ( h_temp, d_temp, (M+2) * (N+2) * sizeof(real_t), hipMemcpyDeviceToHost );
            domain_save ( iteration );
        }
    }

    gettimeofday ( &t_end, NULL );
    printf ( "Total elapsed time: %lf seconds\n",
            WALLTIME(t_end) - WALLTIME(t_start)
            );


    domain_finalize();

    exit ( EXIT_SUCCESS );
}


// 4: Make time_step() a cooperative CUDA kernel
//         where one thread is responsible for one grid point.
void __global__
time_step_kernel (real_t *temp, real_t *thermal_diffusivity, int_t M, int_t N, real_t dt )
{
    //define global thread index
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    cg::grid_group grid = cg::this_grid();

    boundary_condition (temp, M, N, x, y );

    if (x > 0 && x < N && y > 0 && y < M && (x + y) % 2 == 0) {
        real_t c, t, b, l, r, K, A, D, new_value;

        c = T(x, y);

        t = T(x - 1, y);
        b = T(x + 1, y);
        l = T(x, y - 1);
        r = T(x, y + 1);

        K = THERMAL_DIFFUSIVITY(x, y);
        A = -K * dt;
        D = 1.0f + 4.0f * K * dt;

        new_value = (c - A * (t + b + l + r)) / D;

        temp[y * (N + 2) + x] = new_value;
    }
    // black-tile threads go here after failing the if check and wait until red tiles are done computing
    grid.sync();
    //after the threads have been synced, the black tiles are computed
    if (x > 0 && x < N && y > 0 && y < M && (x + y) % 2 == 1) {
        real_t c, t, b, l, r, K, A, D, new_value;

        c = T(x, y);

        t = T(x - 1, y);
        b = T(x + 1, y);
        l = T(x, y - 1);
        r = T(x, y + 1);

        K = THERMAL_DIFFUSIVITY(x, y);
        A = -K * dt;
        D = 1.0f + 4.0f * K * dt;

        new_value = (c - A * (t + b + l + r)) / D;

        temp[y * (N + 2) + x] = new_value;
    }

    grid.sync();
}


//5: Make boundary_condition() a device function and
//         call it from the time_step-kernel.
//         Chose appropriate threads to set the boundary values.
void __device__
boundary_condition ( real_t *temp, int_t M, int_t N, int x, int y )
{
    //Apply bountry conditions if the provided global index (x, y) shows that the thread is a boundary thread.
    if ( x == 1 ) T(x - 1, y) = T(x + 1, y);
    if ( y == 1 ) T(x, y - 1) = T(x, y + 1);
    if ( x == N ) T(x + 1, y) = T(x - 1, y);
    if ( y == M ) T(x, y + 1) = T(x, y - 1);
}


void
domain_init ( void )
{
    h_temp = (real_t*) malloc ( (M+2)*(N+2) * sizeof(real_t) );
    h_thermal_diffusivity = (real_t*) malloc ( (M+2)*(N+2) * sizeof(real_t) );

    //2: Allocate device memory.
    hipMalloc ( (void**) &d_temp, (M+2) * (N+2) * sizeof(real_t));
    hipMalloc ( (void**) &d_thermal_diffusivity, (M+2) * (N+2) * sizeof(real_t));

    dt = 0.1;

    for ( int_t y = 1; y <= M; y++ )
    {
        for ( int_t x = 1; x <= N; x++ )
        {
            real_t temperature = 30 + 30 * sin((x + y) / 20.0);
            real_t diffusivity = 0.05 + (30 + 30 * sin((N - x + y) / 20.0)) / 605.0;

            h_temp[ y * (N+2) + x] = temperature;
            h_thermal_diffusivity[ y * (N+2) + x ] = diffusivity;
        }
    }

    //3: Copy data from host to device.
    hipMemcpy ( d_temp,
		 h_temp,
		 (M+2) * (N+2) * sizeof(real_t),
		 hipMemcpyHostToDevice
    );

    hipMemcpy ( d_thermal_diffusivity,
		 h_thermal_diffusivity,
		 (M+2) * (N+2) * sizeof(real_t),
		 hipMemcpyHostToDevice
    );
}


void
domain_save ( int_t iteration )
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset ( filename, 0, 256*sizeof(char) );
    sprintf ( filename, "data/%.5ld.bin", index );

    FILE *out = fopen ( filename, "wb" );
    if ( ! out )
    {
        fprintf(stderr, "Failed to open file: %s\n", filename);
        exit(1);
    }
    fwrite( h_temp, sizeof(real_t), (N + 2) * (M + 2), out );
    fclose ( out );
}


void
domain_finalize ( void )
{
    free ( h_temp );
    free ( h_thermal_diffusivity );

    //8: Free device memory.
    hipFree ( d_temp );
    hipFree ( d_thermal_diffusivity );
}
